#include <iostream>
#include <hip/hip_runtime.h>

#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
        file << ":" << line << " '" << func << "' \n";
        hipDeviceReset();
        exit(99);
    }
}

__global__ void render(float *fb, int max_x, int max_y) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    if ((i >= max_x) || (j >= max_y)) return;

    int pixel_index = j * max_x * 3 + i * 3;
    fb[pixel_index + 0] = float(i) / max_x;
    fb[pixel_index + 1] = float(j) / max_y;
    fb[pixel_index + 2] = 0.2;
}

int main() {
    int nx = 200;
    int ny = 100;
    int num_pixels = nx * ny;
    size_t fb_size = 3 * num_pixels * sizeof(float);

    // Allocate Unified Memory
    float *fb;
    checkCudaErrors(hipMallocManaged((void **)&fb, fb_size));

    // Launch the kernel
    int tx = 8;
    int ty = 8;
    dim3 blocks((nx + tx - 1) / tx, (ny + ty - 1) / ty);
    dim3 threads(tx, ty);
    render<<<blocks, threads>>>(fb, nx, ny);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // Output the image
    std::cout << "P3\n" << nx << " " << ny << "\n255\n";
    for (int j = ny - 1; j >= 0; j--) {
        for (int i = 0; i < nx; i++) {
            size_t pixel_index = j * 3 * nx + i * 3;
            float r = fb[pixel_index + 0];
            float g = fb[pixel_index + 1];
            float b = fb[pixel_index + 2];
            int ir = int(255.99 * r);
            int ig = int(255.99 * g);
            int ib = int(255.99 * b);
            std::cout << ir << " " << ig << " " << ib << "\n";
        }
    }

    // Free memory
    checkCudaErrors(hipFree(fb));
    return 0;
}